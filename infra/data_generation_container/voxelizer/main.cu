#include "hip/hip_runtime.h"
#include "includes/CompFab.h"
#include "math.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "includes/cuda_math.h"

#include <iostream>
#include <string>
#include <sstream>
#include "stdio.h"
#include <vector>

#define RANDOM_SEEDS 1000
#define EPSILONF 0.000001
#define E_PI 3.1415926535897932384626433832795028841971693993751058209749445923078164062

// check cuda calls for errors
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// generates a random float between 0 and 1
__device__ float generate( hiprandState* globalState , int ind) 
{
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState; 
    return RANDOM;
}
// set up random seed buffer
__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
} 


__device__ bool inside(unsigned int numIntersections, bool double_thick) {
	// if (double_thick && numIntersections % 2 == 0) return (numIntersections / 2) % 2 == 1;
	if (double_thick) return (numIntersections / 2) % 2 == 1;
	return numIntersections % 2 == 1;
}

// adapted from: https://en.wikipedia.org/wiki/M%C3%B6ller%E2%80%93Trumbore_intersection_algorithm
__device__ bool intersects(CompFab::Triangle &triangle, float3 dir, float3 pos) {
	float3 V1 = {triangle.m_v1.m_x, triangle.m_v1.m_y, triangle.m_v1.m_z};
	float3 V2 = {triangle.m_v2.m_x, triangle.m_v2.m_y, triangle.m_v2.m_z};
	float3 V3 = {triangle.m_v3.m_x, triangle.m_v3.m_y, triangle.m_v3.m_z};

	//Find vectors for two edges sharing V1
	float3 e1 = V2 - V1;
	float3 e2 = V3 - V1;
	
	// //Begin calculating determinant - also used to calculate u parameter
	float3 P = cross(dir, e2);

	//if determinant is near zero, ray lies in plane of triangle
	float det = dot(e1, P);
	
	//NOT CULLING
	if(det > -EPSILONF && det < EPSILONF) return false;
	float inv_det = 1.f / det;

	// calculate distance from V1 to ray origin
	float3 T = pos - V1;
	//Calculate u parameter and test bound
	float u = dot(T, P) * inv_det;
	//The intersection lies outside of the triangle
	if(u < 0.f || u > 1.f) return false;

	//Prepare to test v parameter
	float3 Q = cross(T, e1);
	//Calculate V parameter and test bound
	float v = dot(dir, Q) * inv_det;
	//The intersection lies outside of the triangle
	if(v < 0.f || u + v  > 1.f) return false;

	float t = dot(e2, Q) * inv_det;

	if(t > EPSILONF) { // ray intersection
		return true;
	}

	// No hit, no win
	return false;
}

// Decides whether or not each voxel is within the given mesh
__global__ void voxelize_kernel( 
	bool* R, CompFab::Triangle* triangles, const int numTriangles, 
	const float spacing, const float3 bottom_left,
	const int w, const int h, const int d, bool double_thick)
{
	// find the position of the voxel
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int zIndex = blockDim.z * blockIdx.z + threadIdx.z;

	// pick an arbitrary sampling direction
	float3 dir = make_float3(1.0, 0.0, 0.0);

	if ( (xIndex < w) && (yIndex < h) && (zIndex < d) )
	{
		// find linearlized index in final boolean array
		unsigned int index_out = zIndex*(w*h)+yIndex*h + xIndex;
		
		// find world space position of the voxel
		float3 pos = make_float3(bottom_left.x + spacing*xIndex,bottom_left.y + spacing*yIndex,bottom_left.z + spacing*zIndex);

		// check if the voxel is inside of the mesh. 
		// if it is inside, then there should be an odd number of 
		// intersections with the surrounding mesh
		unsigned int intersections = 0;
		for (int i = 0; i < numTriangles; ++i)
			if (intersects(triangles[i], dir, pos))
				intersections += 1;

		// store answer
		R[index_out] = inside(intersections, double_thick);
	}
}


// Decides whether or not each voxel is within the given partially un-closed mesh
// checks a variety of directions and picks most common belief
__global__ void voxelize_kernel_open_mesh( 
	// triangles of the mesh being voxelized
	bool* R, CompFab::Triangle* triangles, const int numTriangles, 
	// information about how large the samples are and where they begin
	const float spacing, const float3 bottom_left,
	// number of voxels
	const int w, const int h, const int d, 
	// sampling information for multiple intersection rays
	const int samples, hiprandState* globalState, bool double_thick
	)
{
	// find the position of the voxel
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int zIndex = blockDim.z * blockIdx.z + threadIdx.z;

	if ( (xIndex < w) && (yIndex < h) && (zIndex < d) )
	{
		// find linearlized index in final boolean array
		unsigned int index_out = zIndex*(w*h)+yIndex*h + xIndex;
		// find world space position of the voxel
		float3 pos = make_float3(bottom_left.x + spacing*xIndex,bottom_left.y + spacing*yIndex,bottom_left.z + spacing*zIndex);
		float3 dir;

		// we will randomly sample 3D space by sending rays in randomized directions
		int votes = 0;
		float theta;
		float z;

		for (int j = 0; j < samples; ++j)
		{
			// compute the random direction. Convert from polar to euclidean to get an even distribution
			theta = generate(globalState, index_out % RANDOM_SEEDS) * 2.f * E_PI;
			z = generate(globalState, index_out % RANDOM_SEEDS) * 2.f - 1.f;

			dir.x = sqrt(1-z*z) * cosf(theta);
			dir.y = sqrt(1-z*z) * sinf(theta);
			dir.z = sqrt(1-z*z) * cosf(theta);

			// check if the voxel is inside of the mesh. 
			// if it is inside, then there should be an odd number of 
			// intersections with the surrounding mesh
			unsigned int intersections = 0;
			for (int i = 0; i < numTriangles; ++i)
				if (intersects(triangles[i], dir, pos)) 
					intersections += 1;
			if (inside(intersections, double_thick)) votes += 1;
		}
		// choose the most popular answer from all of the randomized samples
		R[index_out] = votes > (samples / 2.f);
	}
}

// voxelize the given mesh with the given resolution and dimensions
void kernel_wrapper(int samples, int w, int h, int d, CompFab::VoxelGrid *g_voxelGrid, std::vector<CompFab::Triangle> triangles, bool double_thick)
{
	int blocksInX = (w+8-1)/8;
	int blocksInY = (h+8-1)/8;
	int blocksInZ = (d+8-1)/8;

	dim3 Dg(blocksInX, blocksInY, blocksInZ);
	dim3 Db(8, 8, 8);

	hiprandState* devStates;
	if (samples > 0) {
		// set up random numbers
		dim3 tpb(RANDOM_SEEDS,1,1);
	    hipMalloc ( &devStates, RANDOM_SEEDS*sizeof( hiprandState ) );
	    // setup seeds
	    setup_kernel <<< 1, tpb >>> ( devStates, time(NULL) );
	}
	
	// set up boolean array on the GPU
	bool *gpu_inside_array;
	gpuErrchk( hipMalloc( (void **)&gpu_inside_array, sizeof(bool) * w * h * d ) );
	gpuErrchk( hipMemcpy( gpu_inside_array, g_voxelGrid->m_insideArray, sizeof(bool) * w * h * d, hipMemcpyHostToDevice ) );

	// set up triangle array on the GPU
	CompFab::Triangle* triangle_array = &triangles[0];
	CompFab::Triangle* gpu_triangle_array;
	gpuErrchk( hipMalloc( (void **)&gpu_triangle_array, sizeof(CompFab::Triangle) * triangles.size() ) );
	gpuErrchk( hipMemcpy( gpu_triangle_array, triangle_array, sizeof(CompFab::Triangle) * triangles.size(), hipMemcpyHostToDevice ) );

	float3 lower_left = make_float3(g_voxelGrid->m_lowerLeft.m_x, g_voxelGrid->m_lowerLeft.m_y, g_voxelGrid->m_lowerLeft.m_z);
		
	if (samples > 0) {
		voxelize_kernel_open_mesh<<<Dg, Db>>>(gpu_inside_array, gpu_triangle_array, triangles.size(), (float) g_voxelGrid->m_spacing, lower_left, w, h, d, samples, devStates, double_thick);
	} else {
		voxelize_kernel<<<Dg, Db>>>(gpu_inside_array, gpu_triangle_array, triangles.size(), (float) g_voxelGrid->m_spacing, lower_left, w, h, d, double_thick);
	}

	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	gpuErrchk( hipMemcpy( g_voxelGrid->m_insideArray, gpu_inside_array, sizeof(bool) * w * h * d, hipMemcpyDeviceToHost ) );

	gpuErrchk( hipFree(gpu_inside_array) );
	gpuErrchk( hipFree(gpu_triangle_array) );
}
